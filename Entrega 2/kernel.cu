#include "hip/hip_runtime.h"
﻿/**
* ARQUITECTURA DE COMPUTADORES
* 2º Grado en Ingenieria Informatica
*
* Entrega 2
*
* Alumno: Rodrigo Pascual Arnaiz y Villar Solla, Alejandro
* Fecha: 16/11/2022
*
*/
///////////////////////////////////////////////////////////////////////////
// includes
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "gpu_bitmap.h"

// estrutura de Color, por no declarar las partes de un color en un array
struct Color {
	float R, G, B, A;
};

// Defines
#define LADO_CUADRADO 512 // Dimension vertical
#define REJILLA_TAMNYO 8 // Bloques a mostrar
#define DIM ( LADO_CUADRADO / REJILLA_TAMNYO ) // Dimension de los bloques
#define BLANCO Color { 255, 255, 255, 0 } // Color BLANCO
#define NEGRO Color { 0, 0, 0, 0 } // Color NEGRO


// GLOBAL: funcion llamada desde el host y ejecutada en el device (generateImage)
/**
* Funcion: generateImage ( GLOBAL )
* Objetivo: Funcion que genera la rejilla de colores blancos y negros 
* La rejilla es de un 8x8
*
* Param: char* imagen -> Bitmap del device
* Return: void
*/

__global__ void generateImage(unsigned char* imagen)
{	
	Color color;

	// coordenada vertical de cada hilo
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	// coordenada horizontal de cada hilo
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	// coordenada global de cada hilo (indice para acceder a la memoria)
	int posicion = x + y * blockDim.x * gridDim.x;
	// cada hilo obtiene la posicion de un pixel
	int pixel = posicion * 4;

	// Obtenemos la posición teniendo encuenta la dimension de cada cuadrado
	// dimension_factor = LADO_CUADRADO / REJILLA_TAMNYO 
	// 512 / 8 = 64   |  dimension_factor = 64
	// posX = CEIL( Posicion X  / dimension_factor )
	// posY = CEIL( Posicion Y  / dimension_factor )
	int posX = (int) ceil( (float)x / DIM );
	int posY = (int) ceil( (float)y / DIM );

	// Sabemos el color según X e Y
	// Si Y es par es = 1, si Y es impar es 1
	// Si X + (Y % 2 ? 1 : 0)
	// Par colorea en BLANCO y Impar en NEGRO.
	if ( (posX + (posY % 2)) % 2 ) {
		color = BLANCO;
	}
	else {
		color = NEGRO;
	}

	// Como son el mismo color, recorremos los tres canales (R,G,B)
	imagen[pixel + 0] = color.R; 
	imagen[pixel + 1] = color.G; 
	imagen[pixel + 2] = color.B; 
	imagen[pixel + 3] = color.A; 


}
// MAIN: rutina principal ejecutada en el host
int main(int argc, char** argv)
{
	// Declaracion del bitmap:
	// Inicializacion de la estructura RenderGPU
	RenderGPU foto(LADO_CUADRADO, LADO_CUADRADO);
	// Tamaño del bitmap en bytes
	size_t size = foto.image_size();
	// Asignacion y reserva de la memoria en el host (framebuffer) 
	unsigned char* host_bitmap = foto.get_ptr();
	// Reserva en el device 
	unsigned char* dev_bitmap;

	hipMalloc((void**)&dev_bitmap, size);
	// Generamos el bitmap:
	// Lanzamos un kernel con bloques de 256 hilos (16x16)
	// y tantos bloques como hagan falta
	dim3 hilosB(16, 16);
	// Calculamos el numero de bloques necesario (un hilo por cada pixel)
	dim3 Nbloques(LADO_CUADRADO / 16, LADO_CUADRADO / 16);
	// Generamos el bitmap 
	generateImage <<<Nbloques, hilosB >>> (dev_bitmap);
	// Recogemos el bitmap desde la GPU para visualizarlo
	hipMemcpy(host_bitmap, dev_bitmap, size, hipMemcpyDeviceToHost);
	// Visualizacion y salida
	printf("\n...pulsa ESC para finalizar...");
	foto.display_and_exit();
	return 0;
}
///////////////////////////////////////////////////////////////////////////
